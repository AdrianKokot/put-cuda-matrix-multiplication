
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <iomanip>

void matrixMultiplicationCPU(const int *A, const int *B, int *C, const int rowsA, const int colsA, const int rowsB, const int colsB)
{
  for (int rowA = 0; rowA < rowsA; rowA++)
  {
    for (int colB = 0; colB < colsB; colB++)
    {
      int sum = 0;
      for (int i = 0; i < colsA; i++)
      {
        sum += A[rowA * colsA + i] * B[i * colsB + colB];
      }
      C[rowA * colsB + colB] = sum;
    }
  }
}

__global__ void matrixMultiplicationGPU(int *A, int *B, int *C, int rowsA, int colsA, int colsB)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int rowA = tid / colsB;
  int colB = tid % colsB;

  if (rowA < rowsA && colB < colsB)
  {
    int sum = 0;
    for (int i = 0; i < colsA; i++)
    {
      sum += A[rowA * colsA + i] * B[i * colsB + colB];
    }
    C[rowA * colsB + colB] = sum;
  }
}

int main(int argc, char *argv[])
{
  if (argc < 2)
  {
    std::cout << "Podaj sciezke do pliku jako parametr wywolania programu." << std::endl;
    return 1;
  }

  std::cout << "| " << std::setw(20) << "Size"
            << " | " << std::setw(15) << "CPU"
            << " | " << std::setw(15) << "GPU"
            << " | " << std::setw(15) << "Speedup"
            << " |" << std::endl;
  std::cout << "| -------------------- | --------------- | --------------- | --------------- |" << std::endl;

  for (int fileNum = 1; fileNum < argc; fileNum++)
  {

    std::string filename = argv[fileNum];

    std::ifstream file(filename);
    if (!file.is_open())
    {
      std::cout << "Nie udalo sie otworzyc pliku." << std::endl;
      return 1;
    }

    int rowsA, colsA, rowsB, colsB;
    file >> rowsA >> colsA;
    int *matrixA = new int[rowsA * colsA];
    for (int i = 0; i < rowsA * colsA; i++)
    {
      file >> matrixA[i];
    }

    file >> rowsB >> colsB;
    int *matrixB = new int[rowsB * colsB];
    for (int i = 0; i < rowsB * colsB; i++)
    {
      file >> matrixB[i];
    }

    if (colsA != rowsB)
    {
      std::cout << "Niepoprawne rozmiary macierzy. Nie mozna wykonac mnozenia." << std::endl;
      std::cout << "ColsA: " << colsA << " RowsA: " << rowsA << " ColsB: " << colsB << " RowsA: " << rowsA << std::endl;

      return 1;
    }

    int *d_A;
    int *d_B;
    int *d_C;
    int sizeA = rowsA * colsA * sizeof(int);
    int sizeB = rowsB * colsB * sizeof(int);
    int sizeC = rowsA * colsB * sizeof(int);

    // CUDA
    auto start_GPU = std::chrono::steady_clock::now();

    hipMalloc((void **)&d_A, sizeA);
    hipMalloc((void **)&d_B, sizeB);
    hipMalloc((void **)&d_C, sizeC);

    hipMemcpy(d_A, matrixA, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matrixB, sizeB, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (rowsA * colsB + threadsPerBlock - 1) / threadsPerBlock;

    matrixMultiplicationGPU<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rowsA, colsA, colsB);

    int *matrixC_GPU = new int[rowsA * colsB];
    hipMemcpy(matrixC_GPU, d_C, sizeC, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    auto end_GPU = std::chrono::steady_clock::now();
    auto executionTime_GPU = std::chrono::duration_cast<std::chrono::nanoseconds>(end_GPU - start_GPU).count() / 1000000000.0;

    // CPU

    auto start_CPU = std::chrono::steady_clock::now();
    int *matrixC_CPU = new int[rowsA * colsB];
    matrixMultiplicationCPU(matrixA, matrixB, matrixC_CPU, rowsA, colsA, rowsB, colsB);
    auto end_CPU = std::chrono::steady_clock::now();
    auto executionTime_CPU = std::chrono::duration_cast<std::chrono::nanoseconds>(end_CPU - start_CPU).count() / 1000000000.0;

    for (int i = 0; i < rowsA * colsB; i++)
    {
      if (matrixC_CPU[i] != matrixC_GPU[i])
      {
        std::cout << "Wynik mnożenia na CPU i GPU różni się" << std::endl;
        break;
      }
    }

    std::cout << "| " << std::setw(20) << std::to_string(rowsA) + "x" + std::to_string(colsB) << " | " << std::setw(15) << std::fixed << std::setprecision(7) << executionTime_CPU << " | " << std::setw(15) << std::fixed << std::setprecision(7) << executionTime_GPU << " | " << std::setw(15) << std::fixed << std::setprecision(7) << executionTime_CPU / executionTime_GPU << " |" << std::endl;

    delete[] matrixC_CPU;
    delete[] matrixC_GPU;
    delete[] matrixA;
    delete[] matrixB;
  }

  return 0;
}